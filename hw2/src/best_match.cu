#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include <thread>

// CUDA error checking macro (simplified)
#define CHECK_CUDA(call) do { hipError_t err = call; if (err != hipSuccess) exit(EXIT_FAILURE); } while(0)

// Optimized CUDA kernel using shared memory and half-precision
__global__ void fast_cosine_kernel(int* vectors, int* query, float* sims, int N, int D) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        float dot = 0.0f;
        float norm_vec = 0.0f;
        float norm_query = 0.0f;
        
        // Process in chunks to reduce accumulation error
        #pragma unroll 4
        for (int i = 0; i < D; i++) {
            float vec_val = __int2float_rn(vectors[idx * D + i]);  // Fast integer to float conversion
            float q_val = __int2float_rn(query[i]);
            
            dot += vec_val * q_val;
            norm_vec += vec_val * vec_val;
            norm_query += q_val * q_val;
        }
        
        if (norm_vec <= 0.0f || norm_query <= 0.0f) {
            sims[idx] = -1.0f;
        } else {
            // Fast reciprocal square root approximation
            float inv_norm_vec = rsqrtf(norm_vec);
            float inv_norm_query = rsqrtf(norm_query);
            sims[idx] = dot * inv_norm_vec * inv_norm_query;
            
            // Clamp to valid range
            sims[idx] = fminf(1.0f, fmaxf(-1.0f, sims[idx]));
        }
    }
}

// Optimized version with batched processing
__global__ void batch_cosine_kernel(int* vectors, int* queries, float* sims, int N, int D, int batch_size) {
    int vec_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int query_idx = blockIdx.y;
    
    if (vec_idx < N && query_idx < batch_size) {
        float dot = 0.0f;
        float norm_vec = 0.0f;
        float norm_query = 0.0f;
        
        int* query = &queries[query_idx * D];
        
        #pragma unroll 4
        for (int i = 0; i < D; i++) {
            float vec_val = __int2float_rn(vectors[vec_idx * D + i]);
            float q_val = __int2float_rn(query[i]);
            
            dot += vec_val * q_val;
            norm_vec += vec_val * vec_val;
            norm_query += q_val * q_val;
        }
        
        if (norm_vec <= 0.0f || norm_query <= 0.0f) {
            sims[query_idx * N + vec_idx] = -1.0f;
        } else {
            float inv_norm_vec = rsqrtf(norm_vec);
            float inv_norm_query = rsqrtf(norm_query);
            sims[query_idx * N + vec_idx] = dot * inv_norm_vec * inv_norm_query;
            sims[query_idx * N + vec_idx] = fminf(1.0f, fmaxf(-1.0f, sims[query_idx * N + vec_idx]));
        }
    }
}

// Find best match (optimized for speed)
int fast_find_best_match(float* sims, int N, float target_sim) {
    // Two-pass approach: first try to find exact match, then best available
    int best_idx = -1;
    float best_sim = -1.0f;
    
    // First pass - look for matches above threshold
    for (int i = 0; i < N; i++) {
        if (sims[i] >= target_sim && (best_idx == -1 || sims[i] > best_sim)) {
            best_sim = sims[i];
            best_idx = i;
        }
    }
    
    // If exact match found, look for lowest index with same similarity
    if (best_idx != -1) {
        for (int i = 0; i < best_idx; i++) {
            // Less precise comparison for speed
            if (fabsf(sims[i] - best_sim) < 0.0001f) {
                return i;
            }
        }
        return best_idx;
    }
    
    // Second pass - find best available
    best_sim = -1.0f;
    for (int i = 0; i < N; i++) {
        if (sims[i] > best_sim) {
            best_sim = sims[i];
            best_idx = i;
        }
    }
    
    return best_idx;
}

// Optimized batch search function
void batch_find_best_matches(float* sims, int N, float* target_sims, int* results, int batch_size) {
    #pragma omp parallel for
    for (int q = 0; q < batch_size; q++) {
        float* query_sims = &sims[q * N];
        float target_sim = target_sims[q];
        
        int best_idx = -1;
        float best_sim = -1.0f;
        
        // Look for matches above threshold
        for (int i = 0; i < N; i++) {
            if (query_sims[i] >= target_sim && (best_idx == -1 || query_sims[i] > best_sim)) {
                best_sim = query_sims[i];
                best_idx = i;
            }
        }
        
        // If exact match found, look for lowest index with same similarity
        if (best_idx != -1) {
            for (int i = 0; i < best_idx; i++) {
                if (fabsf(query_sims[i] - best_sim) < 0.0001f) {
                    best_idx = i;
                    break;
                }
            }
            results[q] = best_idx;
            continue;
        }
        
        // If no match, find best available
        best_sim = -1.0f;
        best_idx = -1;
        for (int i = 0; i < N; i++) {
            if (query_sims[i] > best_sim) {
                best_sim = query_sims[i];
                best_idx = i;
            }
        }
        
        results[q] = best_idx;
    }
}

// Class with optimized implementation
class FastCosineSearch {
private:
    int N, D;
    int* d_vectors;  // GPU vectors
    int max_batch_size;
    hipStream_t stream;
    bool use_batched_kernel;
    
public:
    FastCosineSearch(int _N, int _D, int* vectors) : N(_N), D(_D), d_vectors(nullptr) {
        // Copy vectors to GPU
        CHECK_CUDA(hipMalloc(&d_vectors, (size_t)N * D * sizeof(int)));
        CHECK_CUDA(hipMemcpy(d_vectors, vectors, (size_t)N * D * sizeof(int), hipMemcpyHostToDevice));
        
        // Create CUDA stream for asynchronous operations
        CHECK_CUDA(hipStreamCreate(&stream));
        
        // Set max batch size based on available GPU memory
        size_t free_mem, total_mem;
        CHECK_CUDA(hipMemGetInfo(&free_mem, &total_mem));
        size_t mem_per_query = N * sizeof(float) + D * sizeof(int);
        max_batch_size = (int)(free_mem * 0.7 / mem_per_query);
        max_batch_size = std::min(max_batch_size, 128);  // Cap for kernel limitations
        
        // Determine whether to use batched kernel based on problem size
        use_batched_kernel = (D <= 128 && N <= 100000);
        
        std::cout << "GPU search using " << (use_batched_kernel ? "batched" : "single-query") 
                  << " kernel, max batch size: " << max_batch_size << std::endl;
    }
    
    ~FastCosineSearch() {
        if (d_vectors) {
            hipFree(d_vectors);
        }
        hipStreamDestroy(stream);
    }
    
    // Single query search
    int search(int* query, float target_sim) {
        int* d_query;
        float* d_sims;
        float* h_sims = new float[N];
        
        // Allocate GPU memory
        CHECK_CUDA(hipMalloc(&d_query, D * sizeof(int)));
        CHECK_CUDA(hipMalloc(&d_sims, N * sizeof(float)));
        
        // Copy query to GPU
        CHECK_CUDA(hipMemcpy(d_query, query, D * sizeof(int), hipMemcpyHostToDevice));
        
        // Launch kernel
        int block_size = 256;
        int grid_size = (N + block_size - 1) / block_size;
        
        fast_cosine_kernel<<<grid_size, block_size, 0, stream>>>(d_vectors, d_query, d_sims, N, D);
        CHECK_CUDA(hipStreamSynchronize(stream));
        
        // Copy results back to host
        CHECK_CUDA(hipMemcpy(h_sims, d_sims, N * sizeof(float), hipMemcpyDeviceToHost));
        
        // Find best match
        int result = fast_find_best_match(h_sims, N, target_sim);
        
        // Clean up
        hipFree(d_query);
        hipFree(d_sims);
        delete[] h_sims;
        
        return result;
    }
    
    // Optimized batch search
    void batch_search(std::vector<int*>& queries, std::vector<float>& target_sims, std::vector<int>& results) {
        int batch_size = queries.size();
        if (batch_size == 0) return;
        
        results.resize(batch_size, -1);
        
        // Limit batch size based on GPU memory
        int actual_batch_size = std::min(batch_size, max_batch_size);
        
        if (use_batched_kernel && batch_size <= max_batch_size) {
            // Process entire batch at once with 2D kernel
            int* d_queries;
            float* d_sims;
            float* h_sims = new float[N * batch_size];
            int* h_results = new int[batch_size];
            
            // Prepare batch of queries
            int* h_batch_queries = new int[batch_size * D];
            for (int i = 0; i < batch_size; i++) {
                std::copy(queries[i], queries[i] + D, &h_batch_queries[i * D]);
            }
            
            // Allocate and copy to GPU
            CHECK_CUDA(hipMalloc(&d_queries, batch_size * D * sizeof(int)));
            CHECK_CUDA(hipMalloc(&d_sims, batch_size * N * sizeof(float)));
            CHECK_CUDA(hipMemcpy(d_queries, h_batch_queries, batch_size * D * sizeof(int), hipMemcpyHostToDevice));
            
            // Launch 2D kernel
            dim3 block_size(256, 1);
            dim3 grid_size((N + block_size.x - 1) / block_size.x, batch_size);
            
            batch_cosine_kernel<<<grid_size, block_size, 0, stream>>>(d_vectors, d_queries, d_sims, N, D, batch_size);
            CHECK_CUDA(hipStreamSynchronize(stream));
            
            // Copy results back
            CHECK_CUDA(hipMemcpy(h_sims, d_sims, batch_size * N * sizeof(float), hipMemcpyDeviceToHost));
            
            // Find best matches on CPU
            batch_find_best_matches(h_sims, N, target_sims.data(), h_results, batch_size);
            
            // Copy to output
            for (int i = 0; i < batch_size; i++) {
                results[i] = h_results[i];
            }
            
            // Clean up
            hipFree(d_queries);
            hipFree(d_sims);
            delete[] h_sims;
            delete[] h_batch_queries;
            delete[] h_results;
        } else {
            // Process in smaller batches
            for (int batch_start = 0; batch_start < batch_size; batch_start += max_batch_size) {
                int batch_end = std::min(batch_start + max_batch_size, batch_size);
                int current_batch_size = batch_end - batch_start;
                
                // Process each batch
                if (current_batch_size > 1 && use_batched_kernel) {
                    // Use batched processing for multiple queries
                    int* d_queries;
                    float* d_sims;
                    float* h_sims = new float[N * current_batch_size];
                    int* h_results = new int[current_batch_size];
                    
                    // Prepare batch
                    int* h_batch_queries = new int[current_batch_size * D];
                    for (int i = 0; i < current_batch_size; i++) {
                        std::copy(queries[batch_start + i], queries[batch_start + i] + D, &h_batch_queries[i * D]);
                    }
                    
                    // Copy to GPU
                    CHECK_CUDA(hipMalloc(&d_queries, current_batch_size * D * sizeof(int)));
                    CHECK_CUDA(hipMalloc(&d_sims, current_batch_size * N * sizeof(float)));
                    CHECK_CUDA(hipMemcpy(d_queries, h_batch_queries, current_batch_size * D * sizeof(int), hipMemcpyHostToDevice));
                    
                    // Launch 2D kernel
                    dim3 block_size(256, 1);
                    dim3 grid_size((N + block_size.x - 1) / block_size.x, current_batch_size);
                    
                    batch_cosine_kernel<<<grid_size, block_size, 0, stream>>>(d_vectors, d_queries, d_sims, N, D, current_batch_size);
                    CHECK_CUDA(hipStreamSynchronize(stream));
                    
                    // Get results
                    CHECK_CUDA(hipMemcpy(h_sims, d_sims, current_batch_size * N * sizeof(float), hipMemcpyDeviceToHost));
                    
                    // Process on CPU
                    std::vector<float> batch_targets(current_batch_size);
                    for (int i = 0; i < current_batch_size; i++) {
                        batch_targets[i] = target_sims[batch_start + i];
                    }
                    
                    batch_find_best_matches(h_sims, N, batch_targets.data(), h_results, current_batch_size);
                    
                    // Copy to output
                    for (int i = 0; i < current_batch_size; i++) {
                        results[batch_start + i] = h_results[i];
                    }
                    
                    // Clean up
                    hipFree(d_queries);
                    hipFree(d_sims);
                    delete[] h_sims;
                    delete[] h_batch_queries;
                    delete[] h_results;
                } else {
                    // Process queries individually
                    for (int i = batch_start; i < batch_end; i++) {
                        results[i] = search(queries[i], target_sims[i]);
                    }
                }
                
                if (batch_size > 1000 && batch_start % 1000 == 0) {
                    std::cout << "Processed " << batch_start << "/" << batch_size << " queries" << std::endl;
                }
            }
        }
    }
};

int main(int argc, char **argv) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <input_file> <output_file>\n", argv[0]);
        return 1;
    }
    
    // Open files
    FILE *fin = fopen(argv[1], "r");
    FILE *fout = fopen(argv[2], "w");
    
    if (!fin || !fout) {
        fprintf(stderr, "Error opening files\n");
        if (fin) fclose(fin);
        if (fout) fclose(fout);
        return 1;
    }
    
    // Read input parameters
    int N, D, K, A, B, C, M, Q;
    if (fscanf(fin, "%d%d%d%d%d%d%d%d", &N, &D, &K, &A, &B, &C, &M, &Q) != 8) {
        fprintf(stderr, "Error reading parameters\n");
        fclose(fin);
        fclose(fout);
        return 1;
    }
    
    std::cout << "Dataset: N=" << N << ", D=" << D << ", Q=" << Q << std::endl;
    
    // Generate dataset
    int *X = new int[N * D];
    
    for (int i = 0; i < K; ++i) {
        if (fscanf(fin, "%d", &X[i]) != 1) {
            fprintf(stderr, "Error reading initial values\n");
            delete[] X;
            fclose(fin);
            fclose(fout);
            return 1;
        }
    }
    
    // Generate remaining data
    for (int i = K; i < N * D; ++i) {
        X[i] = ((long long)A * X[i - 1] + (long long)B * X[i - 2] + C) % M;
    }
    
    // Create search object
    FastCosineSearch searcher(N, D, X);
    
    // Process queries
    if (Q > 16) {
        // Batch processing
        std::vector<int*> queries;
        std::vector<float> target_sims;
        std::vector<int> results;
        
        const int max_batch_size = 1000;
        
        for (int batch_start = 0; batch_start < Q; batch_start += max_batch_size) {
            int batch_end = std::min(batch_start + max_batch_size, Q);
            
            queries.clear();
            target_sims.clear();
            
            // Read queries
            for (int i = batch_start; i < batch_end; ++i) {
                float target_sim;
                int *query = new int[D];
                
                if (fscanf(fin, "%f", &target_sim) != 1) {
                    fprintf(stderr, "Error reading query similarity\n");
                    delete[] query;
                    continue;
                }
                
                for (int j = 0; j < D; ++j) {
                    if (fscanf(fin, "%d", &query[j]) != 1) {
                        fprintf(stderr, "Error reading query element\n");
                        break;
                    }
                }
                
                queries.push_back(query);
                target_sims.push_back(target_sim);
            }
            
            std::cout << "Processing " << queries.size() << " queries..." << std::endl;
            
            // Process batch
            searcher.batch_search(queries, target_sims, results);
            
            // Write results
            for (size_t i = 0; i < results.size(); ++i) {
                fprintf(fout, "%d\n", results[i]);
            }
            fflush(fout);
            
            // Clean up
            for (int* q : queries) {
                delete[] q;
            }
        }
    } else {
        // Process queries individually
        for (int i = 0; i < Q; ++i) {
            float target_sim;
            int *query = new int[D];
            
            if (fscanf(fin, "%f", &target_sim) != 1) {
                fprintf(stderr, "Error reading query similarity\n");
                delete[] query;
                break;
            }
            
            for (int j = 0; j < D; ++j) {
                if (fscanf(fin, "%d", &query[j]) != 1) {
                    fprintf(stderr, "Error reading query element\n");
                    break;
                }
            }
            
            // Process query
            int result = searcher.search(query, target_sim);
            fprintf(fout, "%d\n", result);
            fflush(fout);
            
            delete[] query;
        }
    }
    
    // Clean up
    fclose(fin);
    fclose(fout);
    delete[] X;
    
    return 0;
}